#include "hip/hip_runtime.h"
#include "sgemm.h"
#include <stdio.h>
#define BLOCK_DIM 16

__global__ void sgemm_shared_kernel(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta) {

    // local thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    // global index
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;

    // global index out of range
    if (col >= N || row >= M) {
        return;
    }

    __shared__ float s_tile_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float s_tile_B[BLOCK_DIM][BLOCK_DIM];

    float sum = 0.f;

    #pragma unroll
    for (int bk = 0; bk < K; bk += BLOCK_DIM) {

        s_tile_A[ty][tx] = A[row * K + bk + tx];    // load A to shared memory
        s_tile_B[ty][tx] = B[(bk + ty) * N + col];  // load B to shared memory

        __syncthreads();

        #pragma unroll
        for (int e = 0; e < BLOCK_DIM; ++e) {
            float a = s_tile_A[ty][e];
            float b = s_tile_B[e][tx];
            sum += a * b;
        }

	    __syncthreads();
    }

    C[row * N + col] = alpha * sum + beta * C[row * N + col];
}

void sgemm_shared(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta) {

    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((N + BLOCK_DIM - 1) / BLOCK_DIM, (M + BLOCK_DIM - 1) / BLOCK_DIM);

    sgemm_shared_kernel << < dimGrid, dimBlock >> > (A, B, C, M, N, K, alpha, beta);
}